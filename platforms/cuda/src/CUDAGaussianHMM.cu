#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <string>
#include "CUDAGaussianHMM.hpp"

#include <hipblas.h>
#include "forward.cu"
#include "backward.cu"
#include "gaussian_likelihood.cu"
#include "posteriors.cu"
#include "expectedtransitions.cu"


#define CudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError()  __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line) {
    if (hipSuccess != err) {
        fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                file, line, hipGetErrorString(err));
            exit(EXIT_FAILURE);
    }
    return;
}
inline void __cudaCheckError( const char *file, const int line ) {
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
                file, line, hipGetErrorString(err));
        exit(-1);
    }
 
    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if(hipSuccess != err) {
        fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                file, line, hipGetErrorString(err));
        exit(-1);
    }
#endif
 
    return;
}


namespace Mixtape {

void cudaMalloc2(void** devicePtr, size_t nbytes) {
    CudaSafeCall(hipMalloc(devicePtr, nbytes));
    CudaSafeCall(hipMemset(*devicePtr, 0x55, nbytes));
}

CUDAGaussianHMM::CUDAGaussianHMM(
    const float** sequences,
    const int n_sequences,
    const int* sequence_lengths,
    const int n_states,
    const int n_features)
    : sequences_(sequences)
    , n_sequences_(n_sequences)
    , n_observations_(0)
    , n_states_(n_states)
    , n_features_(n_features)
    , d_fwdlattice_(NULL)
    , d_bwdlattice_(NULL)
    , d_posteriors_(NULL)
    , d_framelogprob_(NULL)
    , d_log_transmat_(NULL)
    , d_log_transmat_T_(NULL)
    , d_means_(NULL)
    , d_variances_(NULL)
    , d_log_startprob_(NULL)
    , d_sequence_lengths_(NULL)
    , d_cum_sequence_lengths_(NULL)
{
    sequence_lengths_.resize(n_sequences);
    cum_sequence_lengths_.resize(n_sequences);
    for (int i = 0; i < n_sequences_; i++) {
        sequence_lengths_[i] = sequence_lengths[i];
        n_observations_ += sequence_lengths[i];
        if (i == 0)
            cum_sequence_lengths_[i] = 0;
        else
            cum_sequence_lengths_[i] = cum_sequence_lengths_[i-1] + sequence_lengths[i-1];
    }
    
    // Arrays of size proportional to the number of observations
    cudaMalloc2((void **) &d_sequences_, n_observations_*n_features_*sizeof(float));
    cudaMalloc2((void **) &d_sequences2_, n_observations_*n_features_*sizeof(float));
    cudaMalloc2((void **) &d_fwdlattice_, n_observations_*n_states_*sizeof(float));
    cudaMalloc2((void **) &d_bwdlattice_, n_observations_*n_states_*sizeof(float));
    cudaMalloc2((void **) &d_posteriors_, n_observations_*n_states_*sizeof(float));
    cudaMalloc2((void **) &d_framelogprob_, n_observations_*n_states_*sizeof(float));
    cudaMalloc2((void **) &d_ones_, n_observations_*sizeof(float));

    // Small data arrays
    cudaMalloc2((void **) &d_log_transmat_, n_states_*n_states_*sizeof(float));
    cudaMalloc2((void **) &d_log_transmat_T_, n_states_*n_states_*sizeof(float));
    cudaMalloc2((void **) &d_means_, n_states_*n_features_*sizeof(float));
    cudaMalloc2((void **) &d_variances_, n_states_*n_features_*sizeof(float));
    cudaMalloc2((void **) &d_log_startprob_, n_states_*sizeof(float));
    cudaMalloc2((void **) &d_sequence_lengths_, n_sequences_*sizeof(float));
    cudaMalloc2((void **) &d_cum_sequence_lengths_, n_sequences_*sizeof(float));

    // Sufficient statistics
    cudaMalloc2((void **) &d_post_, n_states_*sizeof(float));
    cudaMalloc2((void **) &d_obs_, n_states_*n_features_*sizeof(float));
    cudaMalloc2((void **) &d_obs_squared_, n_states_*n_features_*sizeof(float));
    cudaMalloc2((void **) &d_transcounts_, n_states_*n_states_*sizeof(float));

    // Sequence data
    for (int i = 0; i < n_sequences_; i++) {
        int n = sequence_lengths_[i]*n_features_;
        int offset = cum_sequence_lengths_[i]*n_features_;
        CudaSafeCall(hipMemcpy(d_sequences_ + offset, sequences_[i], n*sizeof(float), hipMemcpyHostToDevice));
    }

    CudaSafeCall(hipMemcpy(d_sequence_lengths_, &sequence_lengths_[0],
                            n_sequences_*sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_cum_sequence_lengths_, &cum_sequence_lengths_[0],
                            n_sequences_*sizeof(float), hipMemcpyHostToDevice));

    hipblasStatus_t status = hipblasCreate((hipblasHandle_t*) &cublas_handle_);
    if (status != HIPBLAS_STATUS_SUCCESS) { exit(EXIT_FAILURE); }
    
    fill<<<1, 256>>>(d_ones_, 1.0, n_observations_);
    square<<<1, 256>>>(d_sequences_, n_observations_*n_features_,
                       d_sequences2_);
    hipDeviceSynchronize();
    CudaCheckError();
}


float CUDAGaussianHMM::computeEStep() {
    gaussian_likelihood<<<1, 32>>>(
        d_sequences_, d_means_, d_variances_, n_sequences_,
        d_sequence_lengths_, d_cum_sequence_lengths_, n_states_,
        n_features_, d_framelogprob_);

    forward4<<<1, 32>>>(
        d_log_transmat_T_, d_log_startprob_, d_framelogprob_,
        d_sequence_lengths_, d_cum_sequence_lengths_, n_sequences_,
        d_fwdlattice_);
    backward4<<<1, 32>>>(
        d_log_transmat_T_, d_log_startprob_, d_framelogprob_,
        d_sequence_lengths_, d_cum_sequence_lengths_, n_sequences_,
        d_bwdlattice_);
    hipDeviceSynchronize();
    posteriors4<<<1, 32>>>(
        d_fwdlattice_, d_bwdlattice_, n_sequences_,
        d_sequence_lengths_, d_cum_sequence_lengths_, d_posteriors_);

    hipDeviceSynchronize();
    CudaCheckError();
    return 1.0;
}

void CUDAGaussianHMM::setMeans(const float* means) {
    CudaSafeCall(hipMemcpy(d_means_, means, n_states_*n_features_*sizeof(float), hipMemcpyHostToDevice));
}

void CUDAGaussianHMM::setVariances(const float* variances) {
    CudaSafeCall(hipMemcpy(d_variances_, variances, n_states_*n_features_*sizeof(float), hipMemcpyHostToDevice));
}

void CUDAGaussianHMM::setTransmat(const float* transmat) {
    std::vector<float> log_transmat(n_states_*n_states_);
    std::vector<float> log_transmat_T(n_states_*n_states_);
    for (int i = 0; i < n_states_; i++)
        for (int j = 0; j < n_states_; j++) {
            log_transmat[i*n_states_ + j] = log(transmat[i*n_states_ + j]);
            log_transmat_T[j*n_states_ +i] = log_transmat[i*n_states_ + j];
        }
    CudaSafeCall(hipMemcpy(d_log_transmat_, &log_transmat[0],     n_states_*n_states_*sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_log_transmat_T_, &log_transmat_T[0], n_states_*n_states_*sizeof(float), hipMemcpyHostToDevice));
}

void CUDAGaussianHMM::setStartProb(const float* startProb) {
    std::vector<float> log_startprob(n_states_);
    for (int i = 0; i < n_states_; i++)
        log_startprob[i] = log(startProb[i]);
    CudaSafeCall(hipMemcpy(d_log_startprob_, &log_startprob[0],
                 n_states_*sizeof(float), hipMemcpyHostToDevice));
}

void CUDAGaussianHMM::getFrameLogProb(float* out) {
    CudaSafeCall(hipMemcpy(out, d_framelogprob_, n_observations_*n_states_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getFwdLattice(float* out) {
    CudaSafeCall(hipMemcpy(out, d_fwdlattice_, n_observations_*n_states_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getBwdLattice(float* out) {
    CudaSafeCall(hipMemcpy(out, d_bwdlattice_, n_observations_*n_states_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getPosteriors(float* out) {
    CudaSafeCall(hipMemcpy(out, d_posteriors_, n_observations_*n_states_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getStatsObs(float* out) {
    CudaSafeCall(hipMemcpy(out, d_obs_, n_states_*n_features_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getStatsObsSquared(float* out) {
    CudaSafeCall(hipMemcpy(out, d_obs_squared_, n_states_*n_features_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getStatsPost(float* out) {
    CudaSafeCall(hipMemcpy(out, d_post_, n_states_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getStatsTransCounts(float* out) {
    CudaSafeCall(hipMemcpy(out, d_transcounts_, n_states_*n_states_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::initializeSufficientStatistics(void) {
    CudaSafeCall(hipMemset(d_obs_, 0, n_states_*n_features_*sizeof(float)));
    CudaSafeCall(hipMemset(d_obs_squared_, 0, n_states_*n_features_*sizeof(float)));
    CudaSafeCall(hipMemset(d_post_, 0, n_states_*sizeof(float)));
    CudaSafeCall(hipMemset(d_transcounts_, 0, n_states_*n_states_*sizeof(float)));
}

void CUDAGaussianHMM::computeSufficientStatistics() {
    float alpha = 1.0f;
    float beta = 1.0f;
    hipblasStatus_t status;
    
    // Compute the sufficient statistics for the mean,
    // \Sum_i p(X_i in state_k) * X_i 
    // MATRIX_MULTIPLY(posteriors.T, obs)
    status = hipblasSgemm(
        (hipblasHandle_t) cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_T,
        n_features_, n_states_, n_observations_, &alpha,
        d_sequences_, n_features_,
        d_posteriors_, n_states_,
        &beta, d_obs_, n_features_);

    if (status != HIPBLAS_STATUS_SUCCESS) { fprintf(stderr, "hipblasSgemm() failed at %s:%i\n", __FILE__, __LINE__); exit(EXIT_FAILURE); }

    // Compute the sufficient statistics for the variance,
    // \Sum_i p(X_i in state_k) * X_i**2 
    // MATRIX_MULTIPLY(posteriors.T, obs**2)
    status = hipblasSgemm(
        (hipblasHandle_t) cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_T,
        n_features_, n_states_, n_observations_, &alpha,
        d_sequences2_, n_features_,
        d_posteriors_, n_states_,
        &beta, d_obs_squared_, n_features_);
    if (status != HIPBLAS_STATUS_SUCCESS) { fprintf(stderr, "hipblasSgemm() failed at %s:%i\n", __FILE__, __LINE__); exit(EXIT_FAILURE); }

    // Compute the normalization constant for the posterior weighted 
    // averages, \Sum_i (P_X_i in state_k)
    status = hipblasSgemm(
        (hipblasHandle_t) cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_T,
        1, n_states_, n_observations_, &alpha,
        d_ones_, 1,
        d_posteriors_, n_states_,
        &beta, d_post_, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) { fprintf(stderr, "hipblasSgemm() failed at %s:%i\n", __FILE__, __LINE__); exit(EXIT_FAILURE); }

    transitioncounts<<<1, 32>>>(
        d_fwdlattice_, d_bwdlattice_, d_log_transmat_, d_framelogprob_,
        n_observations_, n_states_, d_transcounts_);
    
    hipDeviceSynchronize();
    CudaCheckError();
}

CUDAGaussianHMM::~CUDAGaussianHMM() {
    CudaSafeCall(hipFree(d_fwdlattice_));
    CudaSafeCall(hipFree(d_bwdlattice_));
    CudaSafeCall(hipFree(d_posteriors_));
    CudaSafeCall(hipFree(d_framelogprob_));
    CudaSafeCall(hipFree(d_log_transmat_));
    CudaSafeCall(hipFree(d_log_transmat_T_));
    CudaSafeCall(hipFree(d_means_));
    CudaSafeCall(hipFree(d_variances_));
    CudaSafeCall(hipFree(d_log_startprob_));
    CudaSafeCall(hipFree(d_sequence_lengths_));
    CudaSafeCall(hipFree(d_cum_sequence_lengths_));
    CudaSafeCall(hipFree(d_ones_));
    CudaSafeCall(hipFree(d_post_));
    CudaSafeCall(hipFree(d_obs_));
    CudaSafeCall(hipFree(d_obs_squared_));
    CudaSafeCall(hipFree(d_transcounts_));
    CudaSafeCall(hipFree(d_sequences_));
    CudaSafeCall(hipFree(d_sequences2_));

    hipblasDestroy((hipblasHandle_t) cublas_handle_);
}



}  // namespace Mixtape
