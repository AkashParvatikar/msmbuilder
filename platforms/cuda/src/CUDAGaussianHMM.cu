#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <string>
#include "CUDAGaussianHMM.hpp"

#include "forward.cu"
#include "backward.cu"
#include "gaussian_likelihood.cu"
#include "posteriors.cu"


#include <hipblas.h>

#define CudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError()  __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line) {
    if (hipSuccess != err) {
        fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                file, line, hipGetErrorString(err));
            exit(EXIT_FAILURE);
    }
    return;
}
inline void __cudaCheckError( const char *file, const int line ) {
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
                file, line, hipGetErrorString(err));
        exit(-1);
    }
 
    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if(hipSuccess != err) {
        fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                file, line, hipGetErrorString(err));
        exit(-1);
    }
#endif
 
    return;
}


namespace Mixtape {
CUDAGaussianHMM::CUDAGaussianHMM(const float* trajectories,
                                 const int n_trajectories,
                                 const int* n_observations,
                                 const int n_states,
                                 const int n_features)
    : trajectories_(trajectories)
    , n_trajectories_(n_trajectories)
    , n_total_observations_(0)
    , n_states_(n_states)
    , n_features_(n_features)
    , d_fwdlattice_(NULL)
    , d_bwdlattice_(NULL)
    , d_posteriors_(NULL)
    , d_framelogprob_(NULL)
    , d_log_transmat_(NULL)
    , d_log_transmat_T_(NULL)
    , d_means_(NULL)
    , d_variances_(NULL)
    , d_log_startprob_(NULL)
    , d_n_observations_(NULL)
    , d_trj_offset_(NULL)
    , d_trajectories_(NULL)
{
    n_total_observations_ = 0;
    n_observations_.resize(n_trajectories);
    trj_offset_.resize(n_trajectories);
    for (int s = 0; s < n_trajectories_; s++) {
        n_total_observations_ += n_observations[s];
        n_observations_[s] = n_observations[s];
    }

    // trj_offset_[s] is the index in the trajectories_ memory
    // blob where the s-th trajectory starts
    trj_offset_[0] = 0;
    trj_offset_.resize(n_trajectories);
    for (int s = 1; s < n_trajectories_; s++)
        trj_offset_[s] = trj_offset_[s-1] + n_observations_[s-1]*n_features;
    
    CudaSafeCall(hipMalloc((void **) &d_trajectories_, n_total_observations_*n_features_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_trajectories2_, n_total_observations_*n_features_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_fwdlattice_, n_total_observations_*n_states_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_bwdlattice_, n_total_observations_*n_states_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_posteriors_, n_total_observations_*n_states_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_framelogprob_, n_total_observations_*n_states_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_log_transmat_, n_states_*n_states_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_log_transmat_T_, n_states_*n_states_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_means_, n_states_*n_features_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_variances_, n_states_*n_features_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_log_startprob_, n_states_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_n_observations_, n_trajectories_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_trj_offset_, n_trajectories_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_ones_,  n_total_observations_*sizeof(float)));
    
    CudaSafeCall(hipMalloc((void **) &d_post_, n_states_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_obs_, n_states_*n_features_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_obs_squared_, n_states_*n_features_*sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_counts_, n_states_*n_states_*sizeof(float)));
    
    CudaSafeCall(hipMemcpy(d_trajectories_, trajectories_, n_total_observations_*n_features_*sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_n_observations_, &n_observations_[0], n_trajectories_*sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_trj_offset_, &trj_offset_[0], n_trajectories_*sizeof(float), hipMemcpyHostToDevice));

    hipblasStatus_t status = hipblasCreate((hipblasHandle_t*) &cublas_handle_);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
    }

    // square the observed trajectories.
    square<<<1, 256>>>(d_trajectories_, n_total_observations_*n_features, d_trajectories2_);
    fill<<<1, 256>>>(d_ones_, 1.0, n_total_observations_);

}


float CUDAGaussianHMM::computeEStep() {
    gaussian_likelihood<<<1, 32>>>(
        d_trajectories_, d_means_, d_variances_, n_trajectories_,
        d_n_observations_, d_trj_offset_, n_states_, n_features_,
        d_framelogprob_);
    forward4<<<1, 32>>>(
        d_log_transmat_T_, d_log_startprob_, d_framelogprob_,
        d_n_observations_, d_trj_offset_, n_trajectories_,
        d_fwdlattice_);
    backward4<<<1, 32>>>(
        d_log_transmat_T_, d_log_startprob_, d_framelogprob_,
        d_n_observations_, d_trj_offset_, n_trajectories_,
        d_bwdlattice_);
    posteriors4<<<1, 32>>>(
        d_fwdlattice_, d_bwdlattice_, n_trajectories_,
        d_n_observations_, d_trj_offset_, d_posteriors_);

    CudaCheckError();
    return 1.0;
}

void CUDAGaussianHMM::setMeans(const float* means) {
    CudaSafeCall(hipMemcpy(d_means_, means, n_states_*n_features_*sizeof(float), hipMemcpyHostToDevice));
}

void CUDAGaussianHMM::setVariances(const float* variances) {
    CudaSafeCall(hipMemcpy(d_variances_, variances, n_states_*n_features_*sizeof(float), hipMemcpyHostToDevice));
}

void CUDAGaussianHMM::setTransmat(const float* transmat) {
    std::vector<float> log_transmat(n_states_*n_states_);
    std::vector<float> log_transmat_T(n_states_*n_states_);
    for (int i = 0; i < n_states_; i++)
        for (int j = 0; j < n_states_; j++) {
            log_transmat[i*n_states_ + j] = log(transmat[i*n_states_ + j]);
            log_transmat_T[j*n_states_ +i] = log_transmat[i*n_states_ + j];
        }
    CudaSafeCall(hipMemcpy(d_log_transmat_, &log_transmat[0],     n_states_*n_states_*sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_log_transmat_T_, &log_transmat_T[0], n_states_*n_states_*sizeof(float), hipMemcpyHostToDevice));
}

void CUDAGaussianHMM::setStartProb(const float* startProb) {
    std::vector<float> log_startprob(n_states_);
    for (int i = 0; i < n_states_; i++)
        log_startprob[i] = log(startProb[i]);
}

void CUDAGaussianHMM::getFrameLogProb(float* out) {
    CudaSafeCall(hipMemcpy(out, d_framelogprob_, n_total_observations_*n_states_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getFwdLattice(float* out) {
    CudaSafeCall(hipMemcpy(out, d_fwdlattice_, n_total_observations_*n_states_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getBwdLattice(float* out) {
    CudaSafeCall(hipMemcpy(out, d_bwdlattice_, n_total_observations_*n_states_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getPosteriors(float* out) {
    CudaSafeCall(hipMemcpy(out, d_posteriors_, n_total_observations_*n_states_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getStatsObs(float* out) {
    CudaSafeCall(hipMemcpy(out, d_obs_, n_states_*n_features_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getStatsObsSquared(float* out) {
    CudaSafeCall(hipMemcpy(out, d_obs_squared_, n_states_*n_features_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getStatsPost(float* out) {
    CudaSafeCall(hipMemcpy(out, d_post_, n_states_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::initializeSufficientStatistics(void) {
    std::vector<float> zeros(n_states_*n_features_, 0.0);
    CudaSafeCall(hipMemcpy(d_obs_,         &zeros[0], n_states_*n_features_*sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_obs_squared_, &zeros[0], n_states_*n_features_*sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_post_,        &zeros[0], n_states_*sizeof(float), hipMemcpyHostToDevice));
}

void CUDAGaussianHMM::computeSufficientStatistics() {
    float alpha = 1.0f;
    float beta = 1.0f;
    hipblasStatus_t status;

    // Compute the sufficient statistics for the mean, \Sum_i p(X_i in state_k) * X_i 
    // MATRIX_MULTIPLY(posteriors.T, obs)
    status = hipblasSgemm(
        (hipblasHandle_t) cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_T,
        n_features_, n_states_, n_total_observations_, &alpha,
        d_trajectories_, n_features_,
        d_posteriors_, n_states_,
        &beta, d_obs_, n_features_);
    if (status != HIPBLAS_STATUS_SUCCESS) { fprintf(stderr, "hipblasSgemm() failed at %s:%i\n", __FILE__, __LINE__); exit(EXIT_FAILURE); }

    // Compute the sufficient statistics for the variance, \Sum_i p(X_i in state_k) * X_i**2 
    // MATRIX_MULTIPLY(posteriors.T, obs**2)
    status = hipblasSgemm(
        (hipblasHandle_t) cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_T,
        n_features_, n_states_, n_total_observations_, &alpha,
        d_trajectories2_, n_features_,
        d_posteriors_, n_states_,
        &beta, d_obs_squared_, n_features_);
    if (status != HIPBLAS_STATUS_SUCCESS) { fprintf(stderr, "hipblasSgemm() failed at %s:%i\n", __FILE__, __LINE__); exit(EXIT_FAILURE); }

    // Compute the normalization constant for the posterior weighted averages, \Sum_i (P_X_i in state_k)
    status = hipblasSgemm(
        (hipblasHandle_t) cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_T,
        1, n_states_, n_total_observations_, &alpha,
        d_ones_, 1,
        d_posteriors_, n_states_,
        &beta, d_post_, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) { fprintf(stderr, "hipblasSgemm() failed at %s:%i\n", __FILE__, __LINE__); exit(EXIT_FAILURE); }

}

CUDAGaussianHMM::~CUDAGaussianHMM() {
    CudaSafeCall(hipFree(d_trajectories_));
    CudaSafeCall(hipFree(d_trajectories2_));
    CudaSafeCall(hipFree(d_fwdlattice_));
    CudaSafeCall(hipFree(d_bwdlattice_));
    CudaSafeCall(hipFree(d_posteriors_));
    CudaSafeCall(hipFree(d_framelogprob_));
    CudaSafeCall(hipFree(d_log_transmat_));
    CudaSafeCall(hipFree(d_log_transmat_T_));
    CudaSafeCall(hipFree(d_means_));
    CudaSafeCall(hipFree(d_variances_));
    CudaSafeCall(hipFree(d_log_startprob_));
    CudaSafeCall(hipFree(d_n_observations_));
    CudaSafeCall(hipFree(d_trj_offset_));
    CudaSafeCall(hipFree(d_ones_));


    CudaSafeCall(hipFree(d_post_));
    CudaSafeCall(hipFree(d_obs_));
    CudaSafeCall(hipFree(d_obs_squared_));
    CudaSafeCall(hipFree(d_counts_));
    hipblasDestroy((hipblasHandle_t) cublas_handle_);
}



}  // namespace Mixtape
