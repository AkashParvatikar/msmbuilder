#include "hip/hip_runtime.h"
/*****************************************************************/
/*    Copyright (c) 2013, Stanford University and the Authors    */
/*    Author: Robert McGibbon <rmcgibbo@gmail.com>               */
/*    Contributors:                                              */
/*                                                               */
/*****************************************************************/

#include <cstdlib>
#include <cmath>
#include <cstdio>
#include <vector>
#include <limits>
#include <string>
#define CUDA_ERROR_CHECK
#include "safecuda.hpp"
#include "MixtapeException.hpp"
#include "CUDAGaussianHMM.hpp"

#include <hipblas.h>
#include "forward.cu"
#include "backward.cu"
#include "gaussian_likelihood.cu"
#include "posteriors.cu"
#include "expectedtransitions.cu"
#include "sufficientstatistics.cu"

//#define USE_CUBLAS
#define NEW_MVN_KERNEL


namespace Mixtape {
CUDAGaussianHMM::CUDAGaussianHMM(const int n_states,
                                 const int n_features)
    : n_sequences_(0)
    , n_states_(n_states)
    // we have special kernels for n_states in [4, 8, 16, 32], and then a
    // generic one for more than 32 states, so if the number of states is less,
    // need to "pad" to the next power in the range.
    , n_pstates_(n_states < 32 ? max(4, static_cast<int>(pow(2, ceil(log(n_states)/log(2))))) : n_states)
    , n_observations_(0)
    , n_features_(n_features)
    , d_fwdlattice_(NULL)
    , d_bwdlattice_(NULL)
    , d_posteriors_(NULL)
    , d_framelogprob_(NULL)
    , d_log_transmat_(NULL)
    , d_log_transmat_T_(NULL)
    , d_means_(NULL)
    , d_variances_(NULL)
    , d_logvariances_(NULL)
    , d_log_startprob_(NULL)
    , d_logprob_(NULL)
    , d_sequence_lengths_(NULL)
    , d_cum_sequence_lengths_(NULL)
    , d_ones_(NULL)
    , d_post_(NULL)
    , d_obs_(NULL)
    , d_obs_squared_(NULL)
    , d_transcounts_(NULL)
    , d_sequences_(NULL)
    , d_sequences2_(NULL)
{
    // Small data arrays
    cudaMalloc2((void **) &d_log_transmat_, n_pstates_*n_pstates_*sizeof(float));
    cudaMalloc2((void **) &d_log_transmat_T_, n_pstates_*n_pstates_*sizeof(float));
    cudaMalloc2((void **) &d_means_, n_pstates_*n_features_*sizeof(float));
    cudaMalloc2((void **) &d_variances_, n_pstates_*n_features_*sizeof(float));
    cudaMalloc2((void **) &d_logvariances_, n_pstates_*n_features_*sizeof(float));
    cudaMalloc2((void **) &d_log_startprob_, n_pstates_*sizeof(float));
    cudaMalloc2((void **) &d_logprob_, sizeof(float));

    // Sufficient statistics
    cudaMalloc2((void **) &d_post_, n_pstates_*sizeof(float));
    cudaMalloc2((void **) &d_obs_, n_pstates_*n_features_*sizeof(float));
    cudaMalloc2((void **) &d_obs_squared_, n_pstates_*n_features_*sizeof(float));
    cudaMalloc2((void **) &d_transcounts_, n_pstates_*n_pstates_*sizeof(float));

    hipblasStatus_t status = hipblasCreate((hipblasHandle_t*) &cublas_handle_);
    if (status != HIPBLAS_STATUS_SUCCESS) { throw MixtapeException("cuBLAS initialization error."); }
    hipDeviceSynchronize();
}

void CUDAGaussianHMM::setSequences(const float** sequences,
                                   const int n_sequences,
                                   const int* sequence_lengths)
{
    n_observations_ = 0;
    n_sequences_ = n_sequences;
    sequence_lengths_.resize(n_sequences);
    cum_sequence_lengths_.resize(n_sequences);
    for (int i = 0; i < n_sequences_; i++) {
        sequence_lengths_[i] = sequence_lengths[i];
        n_observations_ += sequence_lengths[i];
        if (i == 0)
            cum_sequence_lengths_[i] = 0;
        else
            cum_sequence_lengths_[i] = cum_sequence_lengths_[i-1] + sequence_lengths[i-1];
    }
    delSequences();

    // Arrays of size proportional to the number of observations
    cudaMalloc2((void **) &d_sequences_, n_observations_*n_features_*sizeof(float));
#ifdef USE_CUBLAS
    cudaMalloc2((void **) &d_sequences2_, n_observations_*n_features_*sizeof(float));
#endif
    cudaMalloc2((void **) &d_fwdlattice_, n_observations_*n_pstates_*sizeof(float));
    cudaMalloc2((void **) &d_bwdlattice_, n_observations_*n_pstates_*sizeof(float));
    cudaMalloc2((void **) &d_posteriors_, n_observations_*n_pstates_*sizeof(float));
    cudaMalloc2((void **) &d_framelogprob_, n_observations_*n_pstates_*sizeof(float));
    cudaMalloc2((void **) &d_ones_, n_observations_*sizeof(float));

    cudaMalloc2((void **) &d_sequence_lengths_, n_sequences_*sizeof(float));
    cudaMalloc2((void **) &d_cum_sequence_lengths_, n_sequences_*sizeof(float));

    // Copy over sequence data
    for (int i = 0; i < n_sequences_; i++) {
        int n = sequence_lengths_[i]*n_features_;
        int offset = cum_sequence_lengths_[i]*n_features_;
        CudaSafeCall(hipMemcpy(d_sequences_ + offset, sequences[i], n*sizeof(float), hipMemcpyHostToDevice));
    }

    CudaSafeCall(hipMemcpy(d_sequence_lengths_, &sequence_lengths_[0],
                            n_sequences_*sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_cum_sequence_lengths_, &cum_sequence_lengths_[0],
                            n_sequences_*sizeof(float), hipMemcpyHostToDevice));
    fill<<<100, 256>>>(d_ones_, 1.0, n_observations_);
#ifdef USE_CUBLAS
    square<<<1, 256>>>(d_sequences_, n_observations_*n_features_, d_sequences2_);
#endif
    hipDeviceSynchronize();
}

void CUDAGaussianHMM::delSequences() {
    if (d_sequences_ != NULL)
        hipFree(d_sequences_);
#ifdef USE_CUBLAS
    if (d_sequences2_ != NULL)
        hipFree(d_sequences2_);
#endif
    if (d_fwdlattice_ != NULL)
        hipFree(d_fwdlattice_);
    if (d_bwdlattice_ != NULL)
        hipFree(d_bwdlattice_);
    if (d_posteriors_ != NULL)
        hipFree(d_posteriors_);
    if (d_framelogprob_ != NULL)
        hipFree(d_framelogprob_);
    if (d_ones_ != NULL)
        hipFree(d_ones_);
    if (d_sequence_lengths_ != NULL)
        hipFree(d_sequence_lengths_);
    if (d_cum_sequence_lengths_ != NULL)
        hipFree(d_cum_sequence_lengths_);
}

float CUDAGaussianHMM::computeEStep() {
    if (d_sequences_ == NULL)
        throw MixtapeException("Sequence data not initialized");

#ifdef NEW_MVN_KERNEL
    fill<<<100, 256>>>(d_framelogprob_, 0.0, n_observations_*n_pstates_);
#else
    gaussian_likelihood<<<n_sequences_, 256>>>(
         d_sequences_, d_means_, d_variances_, n_sequences_,
         d_sequence_lengths_, d_cum_sequence_lengths_, n_pstates_,
         n_features_, d_framelogprob_);
#endif
    hipDeviceSynchronize();


    if (n_pstates_ == 4) {
#ifdef NEW_MVN_KERNEL
        log_diag_mvn_likelihood<4,16><<<256, 64>>>(
            d_sequences_, d_means_,  d_variances_, d_logvariances_,
            n_observations_, n_pstates_, n_features_, d_framelogprob_);
        hipDeviceSynchronize();
#endif
        forward4<<<max(1, n_sequences_/8), 256>>>(
            d_log_transmat_T_, d_log_startprob_, d_framelogprob_,
            d_sequence_lengths_, d_cum_sequence_lengths_, n_sequences_,
            d_fwdlattice_);
        backward4<<<max(1, n_sequences_/8), 256>>>(
            d_log_transmat_, d_log_startprob_, d_framelogprob_,
            d_sequence_lengths_, d_cum_sequence_lengths_, n_sequences_,
            d_bwdlattice_);
        hipDeviceSynchronize();

        posteriors<4><<<max(1, n_sequences_/8), 256>>>(
            d_fwdlattice_, d_bwdlattice_, n_sequences_,
            d_sequence_lengths_, d_cum_sequence_lengths_, d_posteriors_);
    }
    else if (n_pstates_ == 8) {
#ifdef NEW_MVN_KERNEL
        log_diag_mvn_likelihood<8,16><<<256, 128>>>(
            d_sequences_, d_means_,  d_variances_, d_logvariances_,
            n_observations_, n_pstates_, n_features_, d_framelogprob_);
        hipDeviceSynchronize();
#endif

        forward8<<<max(1, n_sequences_/8) , 256>>>(
            d_log_transmat_T_, d_log_startprob_, d_framelogprob_,
            d_sequence_lengths_, d_cum_sequence_lengths_, n_sequences_,
            d_fwdlattice_);
        backward8<<<max(1, n_sequences_/8), 256>>>(
            d_log_transmat_, d_log_startprob_, d_framelogprob_,
            d_sequence_lengths_, d_cum_sequence_lengths_, n_sequences_,
            d_bwdlattice_);
        hipDeviceSynchronize();

        posteriors<8><<<max(1, n_sequences_/8), 256>>>(
            d_fwdlattice_, d_bwdlattice_, n_sequences_,
            d_sequence_lengths_, d_cum_sequence_lengths_, d_posteriors_);
    } else if (n_pstates_ == 16) {
#ifdef NEW_MVN_KERNEL
        log_diag_mvn_likelihood<8,16><<<256, 128>>>(
            d_sequences_, d_means_,  d_variances_, d_logvariances_,
            n_observations_, n_pstates_, n_features_, d_framelogprob_);
        hipDeviceSynchronize();
#endif

        forward16<<<max(1, n_sequences_/8), 256>>>(
            d_log_transmat_T_, d_log_startprob_, d_framelogprob_,
            d_sequence_lengths_, d_cum_sequence_lengths_, n_sequences_,
            d_fwdlattice_);
        backward16<<<max(1, n_sequences_/8), 256>>>(
            d_log_transmat_, d_log_startprob_, d_framelogprob_,
            d_sequence_lengths_, d_cum_sequence_lengths_, n_sequences_,
            d_bwdlattice_);

        hipDeviceSynchronize();
        posteriors<16><<<max(1, n_sequences_/8), 256>>>(
            d_fwdlattice_, d_bwdlattice_, n_sequences_,
            d_sequence_lengths_, d_cum_sequence_lengths_, d_posteriors_);
    } else if (n_pstates_ == 32) {
#ifdef NEW_MVN_KERNEL
        log_diag_mvn_likelihood<8,16><<<256, 128>>>(
            d_sequences_, d_means_,  d_variances_, d_logvariances_,
            n_observations_, n_pstates_, n_features_, d_framelogprob_);
        hipDeviceSynchronize();
#endif
        forward32<<<max(1, n_sequences_/8), 256>>>(
            d_log_transmat_T_, d_log_startprob_, d_framelogprob_,
            d_sequence_lengths_, d_cum_sequence_lengths_, n_sequences_,
            d_fwdlattice_);
        backward32<<<max(1, n_sequences_/8), 256>>>(
            d_log_transmat_, d_log_startprob_, d_framelogprob_,
            d_sequence_lengths_, d_cum_sequence_lengths_, n_sequences_,
            d_bwdlattice_);

        hipDeviceSynchronize();
        posteriors<32><<<max(1, n_sequences_/8), 256>>>(
            d_fwdlattice_, d_bwdlattice_, n_sequences_,
            d_sequence_lengths_, d_cum_sequence_lengths_, d_posteriors_);

    } else {
        throw MixtapeException("n_states > 32 is not implemented yet");
    }

    hipDeviceSynchronize();
    CudaCheckError();
    return 1.0;
}

void CUDAGaussianHMM::setMeans(const float* means) {
    if (n_pstates_ != n_states_)
        // we need to fill some values in for the means corresponding to the
        // padding states. Otherwise, nans propagate through the system
        fill<<<1, 256>>>(d_means_ + n_states_*n_features_,
                         std::numeric_limits<float>::max(),
                         (n_pstates_-n_states_) * n_features_);
    CudaSafeCall(hipMemcpy(d_means_, means, n_states_*n_features_*sizeof(float), hipMemcpyHostToDevice));
    hipDeviceSynchronize();
}

void CUDAGaussianHMM::getMeans(float* out) {
    CudaSafeCall(hipMemcpy(out, d_means_, n_states_*n_features_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::setVariances(const float* variances) {
    std::vector<float> vars(n_pstates_*n_features_, -1.0);
    std::vector<float> logvars(n_pstates_*n_features_, 1.0);

    for (int i = 0; i < n_states_*n_features_; i++) {
        vars[i] = variances[i];
        logvars[i] = log(variances[i]);
    }
    CudaSafeCall(hipMemcpy(d_variances_, &vars[0], n_pstates_*n_features_*sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_logvariances_, &logvars[0], n_pstates_*n_features_*sizeof(float), hipMemcpyHostToDevice));

    hipDeviceSynchronize();
}

void CUDAGaussianHMM::getVariances(float* out) {
    CudaSafeCall(hipMemcpy(out, d_variances_, n_states_*n_features_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::setTransmat(const float* transmat) {
    std::vector<float> log_transmat(n_pstates_*n_pstates_);
    std::vector<float> log_transmat_T(n_pstates_*n_pstates_);
    for (int i = 0; i < n_states_; i++)
        for (int j = 0; j < n_states_; j++)
            log_transmat[i*n_pstates_ + j] = log(transmat[i*n_states_ + j]);

    for (int i = n_states_; i < n_pstates_; i++)
        for (int j = n_states_; j < n_pstates_; j++)
            log_transmat[i*n_pstates_ + j] = -std::numeric_limits<float>::max();

    for (int i = 0; i < n_pstates_; i++)
        for (int j = 0; j < n_pstates_; j++)
            log_transmat_T[j*n_pstates_ +i] = log_transmat[i*n_pstates_ + j];

    CudaSafeCall(hipMemcpy(d_log_transmat_, &log_transmat[0],     n_pstates_*n_pstates_*sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_log_transmat_T_, &log_transmat_T[0], n_pstates_*n_pstates_*sizeof(float), hipMemcpyHostToDevice));
}

void CUDAGaussianHMM::getTransmat(float* out) {
    std::vector<float> log_transmat(n_pstates_*n_pstates_);
    CudaSafeCall(hipMemcpy(&log_transmat[0], d_log_transmat_, n_pstates_*n_pstates_*sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < n_states_; i++)
        for (int j = 0; j < n_states_; j++)
            out[i*n_states_ + j] = exp(log_transmat[i*n_pstates_ + j]);
}

void CUDAGaussianHMM::setStartProb(const float* startProb) {
    std::vector<float> log_startprob(n_pstates_);
    for (int i = 0; i < n_states_; i++)
        log_startprob[i] = log(startProb[i]);
    for (int i = n_states_; i < n_pstates_; i++)
        log_startprob[i] = -std::numeric_limits<float>::max();

    CudaSafeCall(hipMemcpy(d_log_startprob_, &log_startprob[0],
                 n_pstates_*sizeof(float), hipMemcpyHostToDevice));
}

void CUDAGaussianHMM::getStartProb(float* out) {
    std::vector<float> log_startprob(n_states_);
    CudaSafeCall(hipMemcpy(&log_startprob[0], d_log_startprob_, n_states_*sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < n_states_; i++)
        out[i] = exp(log_startprob[i]);
}

void CUDAGaussianHMM::getFrameLogProb(float* out) {
    std::vector<float> buf(n_observations_*n_pstates_);
    CudaSafeCall(hipMemcpy(&buf[0], d_framelogprob_, n_observations_*n_pstates_*sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < n_observations_; i++)
        for (int j = 0; j < n_states_; j++)
            out[i*n_states_ + j] = buf[i*n_pstates_ + j];
}

void CUDAGaussianHMM::getFwdLattice(float* out) {
    std::vector<float> buf(n_observations_*n_pstates_);
    CudaSafeCall(hipMemcpy(&buf[0], d_fwdlattice_, n_observations_*n_pstates_*sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < n_observations_; i++)
        for (int j = 0; j < n_states_; j++)
            out[i*n_states_ + j] = buf[i*n_pstates_ + j];
}

void CUDAGaussianHMM::getBwdLattice(float* out) {
    std::vector<float> buf(n_observations_*n_pstates_);
    CudaSafeCall(hipMemcpy(&buf[0], d_bwdlattice_, n_observations_*n_pstates_*sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < n_observations_; i++)
        for (int j = 0; j < n_states_; j++)
            out[i*n_states_ + j] = buf[i*n_pstates_ + j];
}

void CUDAGaussianHMM::getPosteriors(float* out) {
    std::vector<float> buf(n_observations_*n_pstates_);
    CudaSafeCall(hipMemcpy(&buf[0], d_posteriors_, n_observations_*n_pstates_*sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < n_observations_; i++)
        for (int j = 0; j < n_states_; j++)
            out[i*n_states_ + j] = buf[i*n_pstates_ + j];
}

void CUDAGaussianHMM::getStatsObs(float* out) {
    // we just avoid fetching the padding states, since all of the valid
    // memory is contiguous in 1d, since n_states is the major axis
    CudaSafeCall(hipMemcpy(out, d_obs_, n_states_*n_features_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getStatsObsSquared(float* out) {
    // we just avoid fetching the padding states, since all of the valid
    // memory is contiguous in 1d, since n_states is the major axis
    CudaSafeCall(hipMemcpy(out, d_obs_squared_, n_states_*n_features_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getStatsPost(float* out) {
    // we just avoid fetching the padding states, since all of the valid
    // memory is contiguous in 1d
    CudaSafeCall(hipMemcpy(out, d_post_, n_states_*sizeof(float), hipMemcpyDeviceToHost));
}

void CUDAGaussianHMM::getStatsTransCounts(float* out) {
    std::vector<float> transcounts(n_pstates_*n_pstates_);
    CudaSafeCall(hipMemcpy(&transcounts[0], d_transcounts_, n_pstates_*n_pstates_*sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < n_states_; i++)
        for (int j = 0; j < n_states_; j++)
            out[i*n_states_ + j] = transcounts[i*n_pstates_ + j];
}

void CUDAGaussianHMM::initializeSufficientStatistics(void) {
    CudaSafeCall(hipMemset(d_obs_, 0, n_pstates_*n_features_*sizeof(float)));
    CudaSafeCall(hipMemset(d_obs_squared_, 0, n_pstates_*n_features_*sizeof(float)));
    CudaSafeCall(hipMemset(d_post_, 0, n_pstates_*sizeof(float)));
    CudaSafeCall(hipMemset(d_transcounts_, 0, n_pstates_*n_pstates_*sizeof(float)));
    CudaSafeCall(hipMemset(d_logprob_, 0, sizeof(float)));
}

float CUDAGaussianHMM::computeSufficientStatistics() {
#ifdef USE_CUBLAS
    float alpha = 1.0f;
    float beta = 1.0f;
    hipblasStatus_t status;

    // Compute the sufficient statistics for the mean,
    // \Sum_i p(X_i in state_k) * X_i
    // MATRIX_MULTIPLY(posteriors.T, obs)
    status = hipblasSgemm(
        (hipblasHandle_t) cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_T,
        n_features_, n_pstates_, n_observations_, &alpha,
        d_sequences_, n_features_,
        d_posteriors_, n_pstates_,
        &beta, d_obs_, n_features_);
    if (status != HIPBLAS_STATUS_SUCCESS) { fprintf(stderr, "hipblasSgemm() failed at %s:%i code=%s\n", __FILE__, __LINE__, _cudaGetErrorEnum(status)); exit(EXIT_FAILURE); }

    // Compute the sufficient statistics for the variance,
    // \Sum_i p(X_i in state_k) * X_i**2
    // MATRIX_MULTIPLY(posteriors.T, obs**2)
    status = hipblasSgemm(
        (hipblasHandle_t) cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_T,
        n_features_, n_pstates_, n_observations_, &alpha,
        d_sequences2_, n_features_,
        d_posteriors_, n_pstates_,
        &beta, d_obs_squared_, n_features_);
    if (status != HIPBLAS_STATUS_SUCCESS) { fprintf(stderr, "hipblasSgemm() failed at %s:%i\n", __FILE__, __LINE__); exit(EXIT_FAILURE); }

    // Compute the normalization constant for the posterior weighted
    // averages, \Sum_i (P_X_i in state_k)
    status = hipblasSgemm(
        (hipblasHandle_t) cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_T,
        1, n_pstates_, n_observations_, &alpha,
        d_ones_, 1,
        d_posteriors_, n_pstates_,
        &beta, d_post_, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) { fprintf(stderr, "hipblasSgemm() failed at %s:%i\n", __FILE__, __LINE__); exit(EXIT_FAILURE); }
#else
    sufficientstatistics<4, 128><<<100, 128>>>(
        d_posteriors_, d_sequences_, n_observations_, n_pstates_, n_features_,
        d_obs_, d_obs_squared_, d_post_);
#endif

    switch (n_pstates_) {
    case 4:
        transitioncounts4_8_16<4, 256><<<max(1, n_sequences_/16), 256>>>(
            d_fwdlattice_, d_bwdlattice_, d_log_transmat_, d_framelogprob_,
            d_sequence_lengths_, d_cum_sequence_lengths_, n_sequences_,
            d_transcounts_, d_logprob_);
        break;
    case 8:
        transitioncounts4_8_16<8, 256><<<max(1, n_sequences_/16), 256>>>(
            d_fwdlattice_, d_bwdlattice_, d_log_transmat_, d_framelogprob_,
            d_sequence_lengths_, d_cum_sequence_lengths_, n_sequences_,
            d_transcounts_, d_logprob_);
        break;
    case 16:
        transitioncounts4_8_16<16, 256><<<max(1, n_sequences_/16), 256>>>(
            d_fwdlattice_, d_bwdlattice_, d_log_transmat_, d_framelogprob_,
            d_sequence_lengths_, d_cum_sequence_lengths_, n_sequences_,
            d_transcounts_, d_logprob_);
        break;
    default:
        throw MixtapeException("NotImplementedError");
    }

    float logprob = 0;
    CudaSafeCall(hipMemcpy(&logprob, d_logprob_, sizeof(float), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    CudaCheckError();
    return logprob;
}

CUDAGaussianHMM::~CUDAGaussianHMM() {
    CudaSafeCall(hipFree(d_fwdlattice_));
    CudaSafeCall(hipFree(d_bwdlattice_));
    CudaSafeCall(hipFree(d_posteriors_));
    CudaSafeCall(hipFree(d_framelogprob_));
    CudaSafeCall(hipFree(d_log_transmat_));
    CudaSafeCall(hipFree(d_log_transmat_T_));
    CudaSafeCall(hipFree(d_means_));
    CudaSafeCall(hipFree(d_variances_));
    CudaSafeCall(hipFree(d_logvariances_));
    CudaSafeCall(hipFree(d_log_startprob_));
    CudaSafeCall(hipFree(d_logprob_));
    CudaSafeCall(hipFree(d_sequence_lengths_));
    CudaSafeCall(hipFree(d_cum_sequence_lengths_));
    CudaSafeCall(hipFree(d_ones_));
    CudaSafeCall(hipFree(d_post_));
    CudaSafeCall(hipFree(d_obs_));
    CudaSafeCall(hipFree(d_obs_squared_));
    CudaSafeCall(hipFree(d_transcounts_));
    CudaSafeCall(hipFree(d_sequences_));
#ifdef USE_CUBLAS
    CudaSafeCall(hipFree(d_sequences2_));
#endif

    hipblasDestroy((hipblasHandle_t) cublas_handle_);
}



}  // namespace Mixtape
