#include "hip/hip_runtime.h"
#include "logsumexp.cuh"
#include "gaussian_likelihood.cuh"
#include <stdlib.h>

__global__ void gaussian_likelihood(
const float* __restrict__ sequences,
const float* __restrict__ means,
const float* __restrict__ variances,
const size_t n_trajs,
const size_t* __restrict__ n_observations,
const size_t* __restrict__ trj_offsets,
const size_t n_states,
const size_t n_features,
float* __restrict__ loglikelihoods)
{
   const unsigned int WARPS_PER_TRAJ = 4;
   const unsigned int WARP_WIDTH = 32;
   const unsigned int FEATURE_WIDTH = ((n_features + WARP_WIDTH - 1) / WARP_WIDTH) * WARP_WIDTH;
   const float log_M_2_PI = 1.8378770664093453f;
   unsigned int gid = blockIdx.x*blockDim.x+threadIdx.x;
   float temp;
  
   while (gid / (WARP_WIDTH*WARPS_PER_TRAJ) < n_trajs) {
       const unsigned int s = gid / (WARP_WIDTH*WARPS_PER_TRAJ);
       const unsigned int lid = gid % 32;

       for (int t = 0; t < n_observations[s]; t++) {
           for (int j = gid / WARP_WIDTH; j < n_states; j += WARPS_PER_TRAJ) {
               float accumulator = 0;
               for (int i = lid; i < FEATURE_WIDTH; i += WARP_WIDTH) {
                   if (i < n_features) {
                       const float mu = means[j*n_features + i];
                       const float sigma2 = variances[j*n_features + i];
                       const float x = sequences[trj_offsets[s] + t*n_features + i];
                       temp = -0.5f*(log_M_2_PI + log(sigma2) + (x-mu)*(x-mu)/sigma2);
                   } else
                       temp = 0;    
                   accumulator += sum<32>(temp);
               }
               if (lid == 0) {
                   loglikelihoods[trj_offsets[s] + t*n_states + j] = accumulator;
               }
           }
       }
       gid += gridDim.x*blockDim.x;
   }
}
