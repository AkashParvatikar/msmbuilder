#include "hip/hip_runtime.h"
#include "logsumexp.cuh"
#include "forward.cuh"
#include <stdlib.h>

__global__ void forward4(
const float* __restrict__ log_transmat_T,
const float* __restrict__ log_startprob,
const float* __restrict__ frame_logprob,
const size_t* __restrict__ n_observations,
const size_t* __restrict__ trj_offsets,
const size_t n_trajs,
float* __restrict__ fwdlattice)
{
    const int n_states = 4;
    unsigned int gid = blockIdx.x*blockDim.x+threadIdx.x;
    float work_buffer;
    unsigned int t;

    while (gid/16 < n_trajs) {
        const unsigned int hid = gid % 16;
        const unsigned int s = gid / 16;

        if (hid < 4)
             fwdlattice[trj_offsets[s] + hid] = log_startprob[hid] + frame_logprob[trj_offsets[s] + hid];

        for (t = 1; t < n_observations[s]; t++) {
            work_buffer = fwdlattice[trj_offsets[s] + (t-1)*n_states + hid%4] + log_transmat_T[hid];
            work_buffer = logsumexp<4>(work_buffer);
            if (hid % 4 == 0)
                fwdlattice[trj_offsets[s] + t*n_states + hid/4] = work_buffer + frame_logprob[trj_offsets[s] + t*n_states + hid/4];
        }
        gid += gridDim.x*blockDim.x;
    }
}


__global__ void forward8(
const float* __restrict__ log_transmat_T,
const float* __restrict__ log_startprob,
const float* __restrict__ frame_logprob,
const size_t* __restrict__ n_observations,
const size_t* __restrict__ trj_offsets,
const size_t n_trajs,
float* __restrict__ fwdlattice)
{
    const int n_states = 8;
    unsigned int gid = blockIdx.x*blockDim.x+threadIdx.x;
    float work_buffer1, work_buffer2;
    unsigned int t;

    while (gid/32 < n_trajs) {
        const unsigned int lid = gid % 32;
        const unsigned int s = gid / 32;
        const int i = lid % 8;
        const int j1 = lid / 8;
        const int j2 = lid / 8 + 4;

        if (lid < 8)
            fwdlattice[trj_offsets[s] + lid] = log_startprob[lid] + frame_logprob[trj_offsets[s] + lid];

        for (t = 1; t < n_observations[s]; t++) {
            work_buffer1 = fwdlattice[trj_offsets[s] + (t-1)*n_states + i] + log_transmat_T[j1*n_states + i];
            work_buffer2 = fwdlattice[trj_offsets[s] + (t-1)*n_states + i] + log_transmat_T[j2*n_states + i];
            work_buffer1 = logsumexp<8>(work_buffer1);
            work_buffer1 = logsumexp<8>(work_buffer2);
            if (lid % 8 == 0) {
                fwdlattice[trj_offsets[s] + t*n_states + j1] = work_buffer1 + frame_logprob[trj_offsets[s] + t*n_states + j1];
                fwdlattice[trj_offsets[s] + t*n_states + j2] = work_buffer2 + frame_logprob[trj_offsets[s] + t*n_states + j2];
            }
        }
        gid += gridDim.x*blockDim.x;
    }
}


__global__ void forward16(
const float* __restrict__ log_transmat_T,
const float* __restrict__ log_startprob,
const float* __restrict__ frame_logprob,
const size_t* __restrict__ n_observations,
const size_t* __restrict__ trj_offsets,
const size_t n_trajs,
float* __restrict__ fwdlattice)
{
    const int n_states = 16;
    unsigned int gid = blockIdx.x*blockDim.x+threadIdx.x;
    float work_buffer1, work_buffer2;
    unsigned int t, j;

    while (gid/32 < n_trajs) {
        const unsigned int lid = gid % 32;
        const unsigned int s = gid / 32;
 
        if (lid < 16)
            fwdlattice[trj_offsets[s] + lid] = log_startprob[lid] + frame_logprob[trj_offsets[s] + lid];

        for (t = 1; t < n_observations[s]; t++) {
              for (j = 0; j < 8; j++) {
                  const int i = lid % 16;
                  const int j1 = j;
                  const int j2 = j + 8;
                  work_buffer1 = fwdlattice[trj_offsets[s] + (t-1)*n_states + i] + log_transmat_T[j1*n_states + i];
                  work_buffer2 = fwdlattice[trj_offsets[s] + (t-1)*n_states + i] + log_transmat_T[j2*n_states + i];
                  work_buffer1 = logsumexp<16>(work_buffer1);
                  work_buffer2 = logsumexp<16>(work_buffer2);

                  if (i % 16 == 0) {
                      fwdlattice[trj_offsets[s] + t*n_states + j1] = work_buffer1 + frame_logprob[trj_offsets[s] + t*n_states + j1];
                      fwdlattice[trj_offsets[s] + t*n_states + j2] = work_buffer2 + frame_logprob[trj_offsets[s] + t*n_states + j2];
                  }
              }
        }
        gid += gridDim.x*blockDim.x;
    }
}

__global__ void forward32(
const float* __restrict__ log_transmat_T,
const float* __restrict__ log_startprob,
const float* __restrict__ frame_logprob,
const size_t* __restrict__ n_observations,
const size_t* __restrict__ trj_offsets,
const size_t n_trajs,
float* __restrict__ fwdlattice)
{
    // WARPS_PER_TRAJ is the number of warps to allocate per trajectory.
    // The warps need to sync with each other at each step in the trajectory,
    // WARPS_PER_TRAJ needs to be small enough for all of the threads to fit
    // in a single block.
    const int n_states = 32;
    const unsigned int WARPS_PER_TRAJ = 4;
    unsigned int gid = blockIdx.x*blockDim.x+threadIdx.x;
    float work_buffer1;
    unsigned int t, j;

    while (gid/(32*WARPS_PER_TRAJ) < n_trajs) {
        const unsigned int lid = gid % 32;
        const unsigned int s = gid / (32*WARPS_PER_TRAJ);
        fwdlattice[trj_offsets[s] + lid] = log_startprob[lid] + frame_logprob[trj_offsets[s] + lid];

        for (t = 1; t < n_observations[s]; t++) {
            for (j = gid/32; j < n_states; j += WARPS_PER_TRAJ) {
                work_buffer1 = fwdlattice[trj_offsets[s] + (t-1)*n_states + lid] + log_transmat_T[j*n_states + lid];
                work_buffer1 = logsumexp<32>(work_buffer1);
                if (lid == 0)
                    fwdlattice[trj_offsets[s] + t*n_states + j] = work_buffer1 + frame_logprob[trj_offsets[s] + t*n_states + j];
            }
            __syncthreads();
        }
        gid += gridDim.x*blockDim.x;
    }
}
