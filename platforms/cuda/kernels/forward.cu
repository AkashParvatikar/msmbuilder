#include "hip/hip_runtime.h"
/*****************************************************************/
/*    Copyright (c) 2013, Stanford University and the Authors    */
/*    Author: Robert McGibbon <rmcgibbo@gmail.com>               */
/*    Contributors:                                              */
/*                                                               */
/*****************************************************************/

#include "logsumexp.cu"
#include <stdlib.h>

__global__ void forward4(
const float* __restrict__ log_transmat_T,
const float* __restrict__ log_startprob,
const float* __restrict__ frame_logprob,
const int* __restrict__ sequence_lengths,
const int* __restrict__ cum_sequence_lengths,
const int n_trajs,
mixed* __restrict__ fwdlattice)
{
    const int n_states = 4;
    unsigned int gid = blockIdx.x*blockDim.x+threadIdx.x;
    mixed work_buffer;
    unsigned int t;

    while (gid/16 < n_trajs) {
        const unsigned int hid = gid % 16;
        const unsigned int s = gid / 16;
        const float* _frame_logprob = frame_logprob + cum_sequence_lengths[s]*n_states;
        mixed* _fwdlattice = fwdlattice + cum_sequence_lengths[s]*n_states;

        if (hid < 4)
            _fwdlattice[hid] = log_startprob[hid] + _frame_logprob[hid];

        for (t = 1; t < sequence_lengths[s]; t++) {
            work_buffer = _fwdlattice[(t-1)*n_states + hid%4] + log_transmat_T[hid];
            work_buffer = logsumexp<mixed, 4>(work_buffer);
            if (hid % 4 == 0)
                _fwdlattice[t*n_states + hid/4] = work_buffer + _frame_logprob[t*n_states + hid/4];
        }
        gid += gridDim.x*blockDim.x;
    }
}

__global__ void forward8(
const float* __restrict__ log_transmat_T,
const float* __restrict__ log_startprob,
const float* __restrict__ frame_logprob,
const int* __restrict__ sequence_lengths,
const int* __restrict__ cum_sequence_lengths,
const int n_trajs,
mixed* __restrict__ fwdlattice)
{
    const int n_states = 8;
    unsigned int gid = blockIdx.x*blockDim.x+threadIdx.x;
    mixed work_buffer1, work_buffer2;
    unsigned int t;

    while (gid/32 < n_trajs) {
        const unsigned int lid = gid % 32;
        const unsigned int s = gid / 32;
        const float* _frame_logprob = frame_logprob + cum_sequence_lengths[s]*n_states;
        mixed* _fwdlattice = fwdlattice + cum_sequence_lengths[s]*n_states;
        const int i = lid % 8;
        const int j1 = lid / 8;
        const int j2 = lid / 8 + 4;

        if (lid < 8)
            _fwdlattice[lid] = log_startprob[lid] + _frame_logprob[lid];

        for (t = 1; t < sequence_lengths[s]; t++) {
            work_buffer1 = _fwdlattice[(t-1)*n_states + i] + log_transmat_T[j1*n_states + i];
            work_buffer2 = _fwdlattice[(t-1)*n_states + i] + log_transmat_T[j2*n_states + i];
            work_buffer1 = logsumexp<mixed, 8>(work_buffer1);
            work_buffer2 = logsumexp<mixed, 8>(work_buffer2);
            if (lid % 8 == 0) {
                _fwdlattice[t*n_states + j1] = work_buffer1 + _frame_logprob[t*n_states + j1];
                _fwdlattice[t*n_states + j2] = work_buffer2 + _frame_logprob[t*n_states + j2];
            }
        }
        gid += gridDim.x*blockDim.x;
    }
}


__global__ void forward16(
const float* __restrict__ log_transmat_T,
const float* __restrict__ log_startprob,
const float* __restrict__ frame_logprob,
const int* __restrict__ sequence_lengths,
const int* __restrict__ cum_sequence_lengths,
const int n_trajs,
mixed* __restrict__ fwdlattice)
{
    const int n_states = 16;
    unsigned int gid = blockIdx.x*blockDim.x+threadIdx.x;
    mixed work_buffer1, work_buffer2;
    unsigned int t, j;

    while (gid/32 < n_trajs) {
        const unsigned int lid = gid % 32;
        const unsigned int s = gid / 32;
        const float* _frame_logprob = frame_logprob + cum_sequence_lengths[s]*n_states;
        mixed* _fwdlattice = fwdlattice + cum_sequence_lengths[s]*n_states;

        if (lid < 16)
            _fwdlattice[lid] = log_startprob[lid] + _frame_logprob[lid];

        for (t = 1; t < sequence_lengths[s]; t++) {
              for (j = 0; j < 8; j++) {
                  const int i = lid % 16;
                  const int j1 = j;
                  const int j2 = j + 8;
                  work_buffer1 = _fwdlattice[ (t-1)*n_states + i] + log_transmat_T[j1*n_states + i];
                  work_buffer2 = _fwdlattice[ (t-1)*n_states + i] + log_transmat_T[j2*n_states + i];
                  work_buffer1 = logsumexp<mixed, 16>(work_buffer1);
                  work_buffer2 = logsumexp<mixed, 16>(work_buffer2);

                  if (i % 16 == 0) {
                      _fwdlattice[ t*n_states + j1] = work_buffer1 + _frame_logprob[t*n_states + j1];
                      _fwdlattice[ t*n_states + j2] = work_buffer2 + _frame_logprob[t*n_states + j2];
                  }
              }
        }
        gid += gridDim.x*blockDim.x;
    }
}

__global__ void forward32(
const float* __restrict__ log_transmat_T,
const float* __restrict__ log_startprob,
const float* __restrict__ frame_logprob,
const int* __restrict__ sequence_lengths,
const int* __restrict__ cum_sequence_lengths,
const int n_trajs,
mixed* __restrict__ fwdlattice)
{
    // WARPS_PER_TRAJ is the number of warps to allocate per trajectory.
    // The warps need to sync with each other at each step in the trajectory,
    // WARPS_PER_TRAJ needs to be small enough for all of the threads to fit
    // in a single block.
    const int n_states = 32;
    const int WARP_WIDTH = 32;
    const unsigned int WARPS_PER_TRAJ = 1;
    unsigned int gid = blockIdx.x*blockDim.x+threadIdx.x;
    mixed work_buffer;

    while (gid / (WARP_WIDTH*WARPS_PER_TRAJ) < n_trajs) {
        const unsigned int lid = gid % WARP_WIDTH;
        const unsigned int s = gid / (WARP_WIDTH);
        //const unsigned int jteam = (gid % (WARP_WIDTH*WARPS_PER_TRAJ)) / WARP_WIDTH;
        const float* _frame_logprob = frame_logprob + cum_sequence_lengths[s]*n_states;
        mixed* _fwdlattice = fwdlattice + cum_sequence_lengths[s]*n_states;

        //if (jteam == 0)
        _fwdlattice[lid] = log_startprob[lid] + _frame_logprob[lid];
        
        for (int t = 1; t < sequence_lengths[s]; t++) {
            //for (int j = jteam; j < n_states; j += WARPS_PER_TRAJ) {
            for (int j = 0; j < n_states; j += WARPS_PER_TRAJ) {
                work_buffer = _fwdlattice[(t-1)*n_states + lid] + log_transmat_T[j*n_states + lid];
                work_buffer = logsumexp<mixed, 32>(work_buffer);
                if (lid == 0) {
                    _fwdlattice[t*n_states + j] =  work_buffer + _frame_logprob[t*n_states + j];
                }
            }
            //__syncthreads();
        }
        gid += gridDim.x*blockDim.x;
    }
}
